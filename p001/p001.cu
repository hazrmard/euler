/*
If we list all the natural numbers below 10 that are multiples of 3 or 5, we get 3, 5, 6 and 9. The sum of these multiples is 23.

Find the sum of all the multiples of 3 or 5 below 1000.
*/

#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

#define TARGET 1000
#define MAXTHREADS 256

using namespace std;

__global__ void sumMultiples(long load, long *dest);

int main() {

	double load = ceil(sqrt(TARGET));
	double totalthreads = ceil(TARGET / load);
	long blocks = ceil(totalthreads / MAXTHREADS);
	int threadsperblock = (totalthreads > MAXTHREADS) ? totalthreads / blocks : totalthreads;

	long *hostdest = new long[blocks];
	long *devdest;
	hipMalloc(&devdest, blocks * sizeof(long));
	hipMemset(devdest, 0, blocks * sizeof(long));

	cout << "Load per thread: " << load << endl;
	cout << "Total threads: " << totalthreads << endl;
	cout << "Blocks: " << blocks << endl;
	cout << "Threads per block: " << threadsperblock << endl;


	sumMultiples <<<blocks, threadsperblock, threadsperblock * sizeof(long)>>> ((long)load, devdest);

	hipMemcpy(hostdest, devdest, blocks * sizeof(long), hipMemcpyDeviceToHost);

	long sum = 0;
	for (int i = 0; i < blocks; ++i) {
		cout << hostdest[i] << endl;
		sum += hostdest[i];
	}

	cout << endl << "Sum: " << sum << endl;

	hipFree(devdest);
	delete[] hostdest;
	return 0;
}


__global__ void sumMultiples(long load, long *dest) {


	extern __shared__ long sums[];
	__syncthreads();

	sums[threadIdx.x] = 0;
	long start = load * (blockIdx.x * blockDim.x + threadIdx.x);
	long end = start + load;
	for (int i = start; i < end && i < TARGET; ++i) {
		if ((i % 3 == 0) || (i % 5 == 0)) {
			sums[threadIdx.x] += i;
		}
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		for (int i = 0; i < blockDim.x; ++i) {
			dest[blockIdx.x] += sums[i];
		}
	}
}